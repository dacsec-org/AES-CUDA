#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <algorithm>
#include <memory>
#include <chrono>
#include <filesystem>


// Macro for error checking
#define CUDA_CHECK(call) cudaCheck(call, __FILE__, __LINE__)

// Function for the above macro
void cudaCheck(hipError_t result, [[maybe_unused]] const char* file, int line) {
    if (result != hipSuccess) {
        std::cerr << "CUDA error at line " << line << ": " << hipGetErrorString(result) << std::endl;
        exit(1);
    }
}

// Function to multiply two 8-bit values
__host__ __device__ uint8_t multiply(uint8_t a, uint8_t b) {
    uint8_t p = 0x00;
    for (int i = 0; i < 8; i++) {
        if (b & 0x01) {
            p ^= a;
        }
        uint8_t msb = a & 0x80;
        a <<= 1;
        if (msb) {
            a ^= 0x1B;  // XOR with the irreducible polynomial x^8 + x^4 + x^3 + x + 1
        }
        b >>= 1;
    }
    return p;
}

// Function to shift rows to the right
__host__ __device__ void shiftRows(uint8_t* state) {
    uint8_t temp;
    // Row 1
    temp = state[1];
    state[1] = state[5];
    state[5] = state[9];
    state[9] = state[13];
    state[13] = temp;
    // Row 2
    temp = state[2];
    state[2] = state[10];
    state[10] = temp;
    temp = state[6];
    state[6] = state[14];
    state[14] = temp;
    // Row 3
    temp = state[3];
    state[3] = state[15];
    state[15] = state[11];
    state[11] = state[7];
    state[7] = temp;
}

// Function to substitute bytes
__host__ __device__ void subBytes(uint8_t* state, const uint8_t* subBytesTable) {
    for (int i = 0; i < 16; i++) {
        int row = (i >> 2) & 0x03;
        int col = i & 0x03;
        state[i] = subBytesTable[row * 16 + col];
    }
}

// Function to add round key
__host__ __device__ void addRoundKey(uint8_t* state, const uint8_t* key, int round) {
    for (int i = 0; i < 16; i++) {
        state[i] ^= key[round * 16 + i];
    }
}

// Function to mix columns
__host__ __device__ void mixColumns(uint8_t* state, const uint8_t* mixColumnsMatrix) {
    for (int col = 0; col < 4; col++) {
        uint8_t s0 = state[col];
        uint8_t s1 = state[col + 4];
        uint8_t s2 = state[col + 8];
        uint8_t s3 = state[col + 12];
        state[col] = multiply(s0, mixColumnsMatrix[0 * 4 + col]) ^
                     multiply(s1, mixColumnsMatrix[1 * 4 + col]) ^
                     multiply(s2, mixColumnsMatrix[2 * 4 + col]) ^
                     multiply(s3, mixColumnsMatrix[3 * 4 + col]);
        state[col + 4] = multiply(s0, mixColumnsMatrix[0 * 4 + (col + 1) % 4]) ^
                         multiply(s1, mixColumnsMatrix[1 * 4 + (col + 1) % 4]) ^
                         multiply(s2, mixColumnsMatrix[2 * 4 + (col + 1) % 4]) ^
                         multiply(s3, mixColumnsMatrix[3 * 4 + (col + 1) % 4]);
        state[col + 8] = multiply(s0, mixColumnsMatrix[0 * 4 + (col + 2) % 4]) ^
                         multiply(s1, mixColumnsMatrix[1 * 4 + (col + 2) % 4]) ^
                         multiply(s2, mixColumnsMatrix[2 * 4 + (col + 2) % 4]) ^
                         multiply(s3, mixColumnsMatrix[3 * 4 + (col + 2) % 4]);
        state[col + 12] = multiply(s0, mixColumnsMatrix[0 * 4 + (col + 3) % 4]) ^
                          multiply(s1, mixColumnsMatrix[1 * 4 + (col + 3) % 4]) ^
                          multiply(s2, mixColumnsMatrix[2 * 4 + (col + 3) % 4]) ^
                          multiply(s3, mixColumnsMatrix[3 * 4 + (col + 3) % 4]);
    }
}

// Function to copy state
__host__ __device__ void copyState(const uint8_t* input, uint8_t* output) {
    for (int i = 0; i < 16; i++) {
        output[i] = input[i];
    }
}

// CUDA kernel for AES encryption
__global__ void aesEncryptKernel(const uint8_t* input, uint8_t* output, const uint8_t* key, const uint8_t* subBytesTable, const uint8_t* mixColumnsMatrix, size_t dataSize) {
    // Variable declarations
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < dataSize / 16) {
        size_t offset = idx * 16;
        uint8_t state[16];
        // Copy input to state
        copyState(&input[offset], state);
        // AES encryption algorithm
        // AddRoundKey for round 0
        addRoundKey(state, key, 0);
        for (int round = 1; round <= 10; round++) {
            // SubBytes and ShiftRows
            subBytes(state, subBytesTable);
            shiftRows(state);
            // MixColumns
            mixColumns(state, mixColumnsMatrix);
            // AddRoundKey
            addRoundKey(state, key, round);
        }
        // SubBytes and ShiftRows for round 11
        subBytes(state, subBytesTable);
        shiftRows(state);
        // AddRoundKey for round 11
        addRoundKey(state, key, 11);
        // Copy state to output
        copyState(state, &output[offset]);
    }
}

// Function to encrypt using AES
void aesEncrypt(const uint8_t* input, uint8_t* output, const uint8_t* key, const uint8_t* subBytesTable, const uint8_t* mixColumnsMatrix, size_t dataSize) {
    // Allocate memory on GPU for input, output, and key
    uint8_t* input_gpu;
    uint8_t* output_gpu;
    uint8_t* key_gpu;
    uint8_t* subBytesTable_gpu;
    uint8_t* mixColumnsMatrix_gpu;
    CUDA_CHECK(hipMalloc((void**)&input_gpu, dataSize * sizeof(uint8_t)));
    CUDA_CHECK(hipMalloc((void**)&output_gpu, dataSize * sizeof(uint8_t)));
    CUDA_CHECK(hipMalloc((void**)&key_gpu, 176 * sizeof(uint8_t)));
    CUDA_CHECK(hipMalloc((void**)&subBytesTable_gpu, 16 * 16 * sizeof(uint8_t)));
    CUDA_CHECK(hipMalloc((void**)&mixColumnsMatrix_gpu, 4 * 4 * sizeof(uint8_t)));
    // Copy input, key, subBytesTable, and mixColumnsMatrix from host to GPU
    CUDA_CHECK(hipMemcpy(input_gpu, input, dataSize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(key_gpu, key, 176, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(subBytesTable_gpu, subBytesTable, 16 * 16, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(mixColumnsMatrix_gpu, mixColumnsMatrix, 4 * 4, hipMemcpyHostToDevice));
    // Set block and grid dimensions
    int numThreadsPerBlock = 256;
    dim3 blockDims(numThreadsPerBlock, 1, 1);
    dim3 gridDims((dataSize + numThreadsPerBlock - 1) / numThreadsPerBlock, 1, 1);
    // Launch AES encryption kernel
    aesEncryptKernel<<<gridDims, blockDims>>>(input_gpu, output_gpu, key_gpu, subBytesTable_gpu, mixColumnsMatrix_gpu, dataSize);
    // Check for kernel launch errors
    CUDA_CHECK(hipGetLastError());
    // Copy output from GPU to host
    CUDA_CHECK(hipMemcpy(output, output_gpu, dataSize, hipMemcpyDeviceToHost));
    // Free GPU memory
    CUDA_CHECK(hipFree(input_gpu));
    CUDA_CHECK(hipFree(output_gpu));
    CUDA_CHECK(hipFree(key_gpu));
    CUDA_CHECK(hipFree(subBytesTable_gpu));
    CUDA_CHECK(hipFree(mixColumnsMatrix_gpu));
}

// Requests and decodes the user's base64 encoded Key
std::vector<uint8_t> userKey() {
    std::string base64Key;
    std::cout << "Enter base64 encoded key (32 bytes '256-bit'): ";
    std::cin >> base64Key;
    std::vector<uint8_t> decodedKey;
    decodedKey.reserve(32);
    const std::string base64Chars = "ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz0123456789+/";
    for (char c : base64Key) {
        if (c == '=') {
            break;
        }
        auto it = std::find(base64Chars.begin(), base64Chars.end(), c);
        if (it != base64Chars.end()) {
            auto index = static_cast<uint8_t>(std::distance(base64Chars.begin(), it));
            decodedKey.push_back(index);
        }
    }
    if (decodedKey.size() != 32) {
        std::cerr << "Invalid key size." << std::endl;
        exit(1);
    }
    return decodedKey;
}

std::pair<std::string, std::vector<uint8_t>> inputLocation() {
    std::string filePath;
    std::cout << "Enter the complete path of the file to be encrypted (e.g., /home/username/path/file.txt): ";
    std::getline(std::cin, filePath);
    // Check if the file exists
    if (!std::filesystem::exists(filePath)) {
        std::cout << "Error: File does not exist." << std::endl;
        return { "", {} };
    }
    // Open the file to retrieve file size and check if it is readable
    std::ifstream inputFile(filePath, std::ios::binary);
    if (!inputFile) {
        std::cerr << "Failed to open input file." << std::endl;
        return { "", {} };
    }
    // Get file size
    inputFile.seekg(0, std::ios::end);
    std::streamsize fileSize = inputFile.tellg();
    inputFile.seekg(0, std::ios::beg);
    // Read file contents into vector
    std::vector<uint8_t> input(fileSize);
    inputFile.read(reinterpret_cast<char*>(input.data()), static_cast<std::streamsize>(input.size()));
    inputFile.close();
    return { filePath, input };
}

// Output location of the encrypted file (defaults to the user's home directory)
std::string outputLocation(const std::string& inputFilePath) {
    std::string homeDir = std::getenv("HOME");
    std::string fileName = std::filesystem::path(inputFilePath).filename().string();
    // Add the '.enc' extension to the file name
    std::string encryptedFileName = fileName + ".enc";
    std::string encryptedFilePath = homeDir + "/" + encryptedFileName;
    return encryptedFilePath;
}

void writeEncryptedFile(const std::string& filePath, const std::vector<uint8_t>& output) {
    std::ofstream outputFile(filePath, std::ios::binary);
    if (outputFile) {
        outputFile.write(reinterpret_cast<const char*>(output.data()), static_cast<std::streamsize>(output.size()));
        outputFile.close();
        std::cout << "Encrypted file written to: " << filePath << std::endl;
    } else {
        std::cerr << "Failed to write encrypted file." << std::endl;
        exit(1);
    }
}

int main() {
    std::pair<std::string, std::vector<uint8_t>> fileData = inputLocation();
    std::string filePath = fileData.first;
    std::vector<uint8_t> input = fileData.second;
    if (filePath.empty() || input.empty()) {
        return 1; // Exit if file input is invalid
    }
    std::vector<uint8_t> decodedKey = userKey();
    size_t dataSize = input.size();
    // Start timer
    auto start = std::chrono::high_resolution_clock::now();
    std::vector<uint8_t> output(dataSize);
    // SubBytes table
    const uint8_t subBytesTable[16][16] = {
        {0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76},
        {0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0},
        {0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15},
        {0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75},
        {0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84},
        {0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf},
        {0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8},
        {0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2},
        {0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73},
        {0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb},
        {0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79},
        {0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08},
        {0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a},
        {0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e},
        {0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf},
        {0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16}
    };
    // MixColumns matrix
    const uint8_t mixColumnsMatrix[4][4] = {
        {0x02, 0x03, 0x01, 0x01},
        {0x01, 0x02, 0x03, 0x01},
        {0x01, 0x01, 0x02, 0x03},
        {0x03, 0x01, 0x01, 0x02}
    };
    aesEncrypt(input.data(), output.data(), decodedKey.data(), &subBytesTable[0][0], &mixColumnsMatrix[0][0], dataSize);
    // Stop timer
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "Encryption time: " << duration << " ms" << std::endl;
    std::string encryptedFilePath = outputLocation(filePath);
    writeEncryptedFile(encryptedFilePath, output);
    return 0;
}