/*
 * The `aesEncryptKernel` is a CUDA kernel that performs AES encryption on the input data.
 * It takes three arguments: `const uint8_t* input`, `uint8_t* output`, and `const uint8_t* key`.
 * The kernel is launched with a grid of `numBlocks` blocks and `blockDims` threads per block.
 * The kernel implements the AES encryption algorithm by applying different transformations to the input data.
 * It consists of multiple rounds, each performing SubBytes, ShiftRows, MixColumns, and AddRoundKey operations.
 * The SubBytes operation replaces each byte with a corresponding value from the SubBytes table.
 * The ShiftRows operation shifts the bytes in each row to the left.
 * The MixColumns operation performs a matrix multiplication on each column of the state.
 * The AddRoundKey operation XORs the state with the round key.
 * After the last round, the SubBytes and ShiftRows operations are performed again, followed by the final AddRoundKey operation.
 * The resulting state is then copied to the output array.
 */
#include <hip/hip_runtime.h>
#include <iostream>

// Macro for checking CUDA errors
#define CUDA_CHECK(call) \
do { \
    hipError_t result = call; \
    if (result != hipSuccess) { \
        std::cerr << "CUDA error at line " << __LINE__ << ": " << hipGetErrorString(result) << std::endl; \
        exit(1); \
    } \
} while(0)

// SubBytes table
__device__ __constant__ uint8_t subBytesTable[16][16] = {
        {0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30,
                0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76},
        {0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad,
                0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0},
        {0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34,
                0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15},
        {0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07,
                0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75},
        {0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52,
                0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84},
        {0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a,
                0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf},
        {0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45,
                0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8},
        {0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc,
                0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2},
        {0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4,
                0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73},
        {0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46,
                0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb},
        {0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2,
                0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79},
        {0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c,
                0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08},
        {0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8,
                0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a},
        {0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61,
                0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e},
        {0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b,
                0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf},
        {0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41,
                0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16}
};

// MixColumns matrix
__device__ __constant__ uint8_t mixColumnsMatrix[4][4] = {
        {0x02, 0x03, 0x01, 0x01},
        {0x01, 0x02, 0x03, 0x01},
        {0x01, 0x01, 0x02, 0x03},
        {0x03, 0x01, 0x01, 0x02}
};

// Function to multiply two 8-bit values
__device__ uint8_t multiply(uint8_t a, uint8_t b) {
    uint8_t result = 0;
    uint8_t mask = 0x01;
    uint8_t p = 0x00;
    for (int i = 0; i < 8; i++) {
        if (b & mask) {
            p ^= a;
        }
        uint8_t msb = a & 0x80;
        a <<= 1;
        if (msb) {
            a ^= 0x1B;  // XOR with the irreducible polynomial x^8 + x^4 + x^3 + x + 1
        }
        mask <<= 1;
    }
    return p;
}

// CUDA kernel for AES encryption
__global__ void aesEncryptKernel(const uint8_t* input, uint8_t* output, const uint8_t* key)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int round;
    uint8_t state[16];

    // Copy input to state
    for (int i = 0; i < 16; i++) {
        state[i] = input[idx * 16 + i];
    }

    // AddRoundKey for round 0
    for (int i = 0; i < 16; i++) {
        state[i] ^= key[i];
    }

    // AES encryption algorithm
    for (round = 1; round <= 9; round++) {
        // SubBytes
        for (unsigned char & i : state) {
            int row = (i >> 4) & 0x0F;
            int col = i & 0x0F;
            i = subBytesTable[row][col];
        }

        // Shift the first Row the right
        uint8_t temp = state[1];
        state[1] = state[5];
        state[5] = state[9];
        state[9] = state[13];
        state[13] = temp;
        // Shift the second Row the right
        temp = state[2];
        state[2] = state[10];
        state[10] = temp;
        // Shift the third Row the right
        temp = state[3];
        state[3] = state[15];
        state[15] = state[11];
        state[11] = state[7];
        state[7] = temp;

        // MixColumns
        for (int col = 0; col < 4; col++) {
            uint8_t s0 = state[col];
            uint8_t s1 = state[col + 4];
            uint8_t s2 = state[col + 8];
            uint8_t s3 = state[col + 12];
            state[col] = multiply(s0, mixColumnsMatrix[0][0]) ^
                         multiply(s1, mixColumnsMatrix[0][1]) ^
                         multiply(s2, mixColumnsMatrix[0][2]) ^
                         multiply(s3, mixColumnsMatrix[0][3]);
            state[col + 4] = multiply(s0, mixColumnsMatrix[1][0]) ^
                             multiply(s1, mixColumnsMatrix[1][1]) ^
                             multiply(s2, mixColumnsMatrix[1][2]) ^
                             multiply(s3, mixColumnsMatrix[1][3]);
            state[col + 8] = multiply(s0, mixColumnsMatrix[2][0]) ^
                             multiply(s1, mixColumnsMatrix[2][1]) ^
                             multiply(s2, mixColumnsMatrix[2][2]) ^
                             multiply(s3, mixColumnsMatrix[2][3]);
            state[col + 12] = multiply(s0, mixColumnsMatrix[3][0]) ^
                              multiply(s1, mixColumnsMatrix[3][1]) ^
                              multiply(s2, mixColumnsMatrix[3][2]) ^
                              multiply(s3, mixColumnsMatrix[3][3]);
        }

        // AddRoundKey
        for (int i = 0; i < 16; i++) {
            state[i] ^= key[round * 16 + i];
        }
    }

    // SubBytes
    for (unsigned char & i : state) {
        int row = (i >> 4) & 0x0F;
        int col = i & 0x0F;
        i = subBytesTable[row][col];
    }

    // shift the 1'st row to the right
    uint8_t temp = state[1];
    state[1] = state[5];
    state[5] = state[9];
    state[9] = state[13];
    state[13] = temp;
    // shift the 2'nd row to the right
    temp = state[2];
    state[2] = state[10];
    state[10] = temp;
    // shift the 3'rd row to the right
    temp = state[3];
    state[3] = state[15];
    state[15] = state[11];
    state[11] = state[7];
    state[7] = temp;

    // AddRoundKey for round 10
    for (int i = 0; i < 16; i++) {
        state[i] ^= key[round * 16 + i];
    }

    // Copy state to output
    for (int i = 0; i < 16; i++) {
        output[idx * 16 + i] = state[i];
    }
}

int main()
{
    // Get CUDA device properties
    hipDeviceProp_t deviceProps{};
    CUDA_CHECK(hipGetDeviceProperties(&deviceProps, 0));

    // Get available global memory
    size_t totalGlobalMem, freeGlobalMem;
    CUDA_CHECK(hipMemGetInfo(&freeGlobalMem, &totalGlobalMem));

    // Calculate data size based on available memory
    const size_t dataSize = freeGlobalMem * 0.8;

    // Calculate number of blocks and threads per block
    const int numBlocks = (dataSize + 15) / 16;
    const int numThreadsPerBlock = 256;

    // Allocate memory on GPU for input, output, and key
    uint8_t* input_gpu;
    CUDA_CHECK(hipMalloc((void**)&input_gpu, dataSize));
    uint8_t* output_gpu;
    CUDA_CHECK(hipMalloc((void**)&output_gpu, dataSize));
    uint8_t* key_gpu;
    CUDA_CHECK(hipMalloc((void**)&key_gpu, 176));

    // Set block and grid dimensions
    dim3 blockDims(numThreadsPerBlock, 1, 1);
    dim3 gridDims(numBlocks, 1, 1);

    // Launch AES encryption kernel
    aesEncryptKernel<<<gridDims, blockDims>>>(input_gpu, output_gpu, key_gpu);

    // Check for kernel launch errors
    CUDA_CHECK(hipGetLastError());

    // Allocate host memory for output
    auto* output = new uint8_t[dataSize];

    // Copy output from GPU to host
    CUDA_CHECK(hipMemcpy(output, output_gpu, dataSize, hipMemcpyDeviceToHost));

    // Free GPU memory
    CUDA_CHECK(hipFree(input_gpu));
    CUDA_CHECK(hipFree(output_gpu));
    CUDA_CHECK(hipFree(key_gpu));

    // Free host memory
    delete[] output;

    return 0;
}